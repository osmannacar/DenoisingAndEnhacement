#include "hip/hip_runtime.h"
#include "bpr.h"
#include "../Utils.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

template<typename TypeSrc>
__device__ void sortArray(TypeSrc *pInput, const int pLenght){
    for(int i = 0; i < pLenght - 1; ++i){
        for(int j = i + 1; j < pLenght; ++j){
            if(pInput[i] > pInput[j]){
                pInput[i] = pInput[i] ^ pInput[j];
                pInput[j] = pInput[i] ^ pInput[j];
                pInput[i] = pInput[i] ^ pInput[j];
            }
        }
    }
}

__global__ void bad_pixel_correction(unsigned short *input, unsigned short *output, const int rows, const int cols, const int kernel) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int fromX = ((x - kernel) > 0 )   ? (x - kernel) : x;
    int toX   = (rows > (x + kernel)) ? (x + kernel) : rows;

    int fromY = ((y - kernel) > 0 )   ? (y - kernel) : y;
    int toY   = (cols > (y + kernel)) ? (y + kernel) : cols;


    unsigned short tBadPixelData = input[ x * cols + y];

    unsigned short neighborhood[50] = {0};
    int count = 0;

    for( int nx = fromX; nx < toX; ++nx){
        for( int ny = fromY; ny < toY; ++ny){
            unsigned short tPixelValue = input[ nx * cols + ny];
            if((tPixelValue != tBadPixelData) && (tPixelValue != 0) && (tPixelValue != USHRT_MAX)){
                neighborhood[count] = tPixelValue;
                ++count;
            }
        }
    }
    if(count > 0){
        sortArray<unsigned short>(neighborhood, count);
        output[ x * cols + y] = neighborhood[count / 2];
        __syncthreads();
    }

}

void badPixelReplacement(const unsigned short *pInput, unsigned short *pOutput, const int rows, const int cols){

    const static int kernel = 3;

    unsigned short *d_input, *d_output;

    GPU_HANDLE_ERROR(hipMalloc(&d_input, rows * cols * sizeof(unsigned short)));
    GPU_HANDLE_ERROR(hipMalloc(&d_output, rows * cols * sizeof(unsigned short)));

    GPU_HANDLE_ERROR(hipMemcpy(d_input, pInput, rows * cols * sizeof(unsigned short), hipMemcpyHostToDevice));

    dim3 blockDim(32, 32);
    dim3 gridDim((rows + blockDim.x - 1) / blockDim.x, (cols + blockDim.y - 1) / blockDim.y);

    bad_pixel_correction<<<gridDim, blockDim>>>(d_input, d_output, rows, cols, kernel);


    GPU_HANDLE_ERROR(hipMemcpy(pOutput, d_output, rows * cols * sizeof(unsigned short), hipMemcpyDeviceToHost));

    GPU_HANDLE_ERROR(hipPeekAtLastError());

    // Wait for all threads to finish
    GPU_HANDLE_ERROR(hipDeviceSynchronize());

    hipFree(d_input);
    hipFree(d_output);
}
