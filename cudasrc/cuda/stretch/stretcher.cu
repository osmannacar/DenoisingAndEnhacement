#include "hip/hip_runtime.h"
#include "stretcher.h"
#include <iostream>
#include <chrono>


#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>

//cuda
#include "../radix_sort/sort.h"
#include "../Utils.h"

template<typename TypeSrc, typename TypeDst>
__global__ void findAvgOfArray(const TypeSrc *pInput, TypeSrc *pOutput, const int pArraySize) {
    int idx = threadIdx.x;

    TypeDst sum = 0.0;
    for (int i = idx; i < pArraySize; i += BLOCK_DIM)
        sum += pInput[i];

    __shared__ TypeDst shArr[BLOCK_DIM];
    shArr[idx] = sum;
    __syncthreads();

    for (int size = BLOCK_DIM/2; size>0; size/=2) { //uniform
        if (idx<size)
            shArr[idx] += shArr[idx+size];
        __syncthreads();
    }
    if (idx == 0)
        *pOutput = shArr[0] / pArraySize;
}

template<typename Type>
__global__ void findMinMaxWithAvgAndRatio(const Type *pInput, const int pArraySize, const Type pAvg, const float pRatio, Type *pMinOutput, Type *pMaxOutput){
    int index = blockDim.x*blockIdx.x + threadIdx.x;
    int shared_index = threadIdx.x;

    __shared__ Type data_shared_min[BLOCK_DIM];
    __shared__ Type data_shared_max[BLOCK_DIM];

    const unsigned short tMaxPixelValue = (pAvg * (1 + pRatio));
    const unsigned short tMinPixelValue = (pAvg * pRatio);

    // check index condition
    if(index < pArraySize){

        if((pInput[index] >= tMinPixelValue) && (pInput[index] <= tMaxPixelValue)){

            data_shared_min[shared_index] = pInput[index]; //pass values from global to shared memory
            __syncthreads();
            data_shared_max[shared_index] = pInput[index]; //pass values from global to shared memory

            for (unsigned int stride = BLOCK_DIM/2; stride > 0; stride >>= 1) {
                if(threadIdx.x <  stride){
                    if(data_shared_max[threadIdx.x] <  data_shared_max[threadIdx.x+stride]) data_shared_max[shared_index] = data_shared_max[shared_index+stride];
                    if(data_shared_min[threadIdx.x] >  data_shared_min[threadIdx.x+stride]) data_shared_min[shared_index] = data_shared_min[shared_index+stride];
                }
                __syncthreads();
            }
            if(threadIdx.x == 0  ){
                atomicMin((unsigned int *)(pMinOutput), (Type)data_shared_min[threadIdx.x ]);
                //min_device =10;
                __syncthreads();
                atomicMax((unsigned int *)(pMaxOutput), (Type)data_shared_max[threadIdx.x ]);
            }

        }
    }else{
        data_shared_min[shared_index] = 9999;
    }
}

template<typename TypeSrc, typename TypeDst>
__global__ void cudaMemCopy(TypeSrc* src, TypeDst* dest, int binDim, size_t cols) {
    int outXind = blockIdx.x * blockDim.x + threadIdx.x;
    int outYind = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate x & y index of input binned pixels corresponding to current output pixel
    int inXstart = outXind * binDim;
    int inYstart = outYind * binDim;
    for (int binY = inYstart; binY < (inYstart + binDim); ++binY) {
        for (int binX = inXstart; binX < (inXstart + binDim); ++binX) {
            int pixel_tid = binY * cols + binX;
            dest[pixel_tid] = src[pixel_tid];
        }
    }
    __syncthreads();
}

/*!
 *working only grayscale image
 */
template<typename T>
__global__ void cudaStretching(T *input, T *output, int binDim, unsigned short cols, unsigned short minPixelval, float ratioOfMaxWithDiff)
{
    int outXind = blockIdx.x * blockDim.x + threadIdx.x;
    int outYind = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate x & y index of input binned pixels corresponding to current output pixel
    int inXstart = outXind * binDim;
    int inYstart = outYind * binDim;

    for (int binY = inYstart; binY < (inYstart + binDim); ++binY) {
        for (int binX = inXstart; binX < (inXstart + binDim); ++binX) {
            int pixel_tid = binY * cols + binX;
            __syncthreads();
            output[pixel_tid] = (input[pixel_tid] - minPixelval) * ratioOfMaxWithDiff;
        }
    }
    __syncthreads();
}


void contrastStretch(const unsigned short * pInput, unsigned short *pOutput, const int pRows, const int pCols)
{
    //2X2 binning
    const static int binDim = 2;

    const int tInputSize = pRows * pCols;

    // Allocate memory in device
    unsigned short *d_input, *d_output, *d_avg_output, *d_min_output, *d_max_output;
//    unsigned int *d_inputInt, *d_sortedOutput;

//    auto start = std::chrono::high_resolution_clock::now();
    GPU_HANDLE_ERROR(hipMalloc<unsigned short>(&d_input, tInputSize * sizeof(unsigned short)));
    GPU_HANDLE_ERROR(hipMalloc<unsigned short>(&d_output, tInputSize * sizeof(unsigned short)));
//    GPU_HANDLE_ERROR(hipMalloc<unsigned int>(&d_inputInt, tInputSize * sizeof(unsigned int)));
//    GPU_HANDLE_ERROR(hipMalloc<unsigned int>(&d_sortedOutput, tInputSize * sizeof(unsigned int)));
    GPU_HANDLE_ERROR(hipMalloc<unsigned short>(&d_avg_output, sizeof(unsigned short)));
    GPU_HANDLE_ERROR(hipMalloc<unsigned short>(&d_min_output, sizeof(unsigned short)));
    GPU_HANDLE_ERROR(hipMalloc<unsigned short>(&d_max_output, sizeof(unsigned short)));

    GPU_HANDLE_ERROR(hipMemset(d_min_output, std::numeric_limits<ushort>::max(), sizeof(unsigned short)));
    GPU_HANDLE_ERROR(hipMemset(d_max_output, std::numeric_limits<ushort>::min(), sizeof(unsigned short)));

//    std::cout<< "Duration Of hipMalloc MicroSec:" << std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start).count() << std::endl;


    // Copy input image to device
//    start = std::chrono::high_resolution_clock::now();
    GPU_HANDLE_ERROR(hipMemcpy(d_input, pInput, tInputSize * sizeof(unsigned short), hipMemcpyHostToDevice));
//    std::cout<< "Duration Of hipMemcpy MicroSec:" << std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start).count() << std::endl;

    // Configure size of block and grid
    const static dim3 block(32, 32);
    const static dim3 grid((pCols + block.x - 1) / block.x, (pRows + block.y - 1) / block.y); // Additional block for rounding up


//    start = std::chrono::high_resolution_clock::now();
    findAvgOfArray<unsigned short, unsigned long long><<<1, BLOCK_DIM>>>(d_input, d_avg_output, tInputSize);
    unsigned short tAvg = 0.0;
    GPU_HANDLE_ERROR(hipMemcpy(&tAvg, d_avg_output, sizeof(unsigned short), hipMemcpyDeviceToHost));

    dim3 numThreads(BLOCK_DIM);
    dim3 numBlocks(tInputSize/BLOCK_DIM + (tInputSize % BLOCK_DIM == 0 ? 0 : 1));

    findMinMaxWithAvgAndRatio<unsigned short><<<numBlocks, numThreads>>>(d_input, tInputSize, tAvg, 0.5, d_min_output, d_max_output);

    ushort tMaxPixel2 = 0;
    ushort tMinPixel2 = 0;

    GPU_HANDLE_ERROR(hipMemcpy(&tMinPixel2, d_min_output, sizeof(unsigned short), hipMemcpyDeviceToHost));
    GPU_HANDLE_ERROR(hipMemcpy(&tMaxPixel2, d_max_output, sizeof(unsigned short), hipMemcpyDeviceToHost));

//    std::cout <<"AVG::" << tAvg << std::endl;
//    std::cout <<"tMinPixel2::" << tMinPixel2 << std::endl;
//    std::cout <<"tMaxPixel2::" << tMaxPixel2 << std::endl;

//    std::cout<< "Duration Of Find SUM, MIN, MAX MicroSec:" << std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start).count() << std::endl;


//    start = std::chrono::high_resolution_clock::now();
//    cudaMemCopy<unsigned short, unsigned int> <<<grid, block>>> (d_input, d_inputInt, binDim, pCols);
//    std::cout<< "Duration Of Copy MicroSec:" << std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start).count() << std::endl;

//    start = std::chrono::high_resolution_clock::now();
//    radix_sort(d_inputInt, d_sortedOutput, tInputSize);
//    std::cout<< "Duration Of Sort MicroSec:" << std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start).count() << std::endl;

    unsigned short min = tMinPixel2, max =tMaxPixel2;
//    GPU_HANDLE_ERROR(hipMemcpy(&min, &d_inputInt[10], sizeof(unsigned short), hipMemcpyDeviceToHost));
//    GPU_HANDLE_ERROR(hipMemcpy(&max, &d_inputInt[tInputSize - 10], sizeof(unsigned short), hipMemcpyDeviceToHost));

//    start = std::chrono::high_resolution_clock::now();
    cudaStretching <unsigned short> <<<grid, block>>> (d_input, d_output, binDim, pCols, min, (UINT16_MAX / (max-min)));
//    std::cout<< "Duration Of Stretching MicroSec:" << std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::high_resolution_clock::now() - start).count() << std::endl;

    GPU_HANDLE_ERROR(hipPeekAtLastError());

    // Wait for all threads to finish
    GPU_HANDLE_ERROR(hipDeviceSynchronize());

    // Copy output image from device back to host (hipMemcpy is a blocking instruction)
    GPU_HANDLE_ERROR(hipMemcpy(pOutput, d_output, tInputSize * sizeof(unsigned short), hipMemcpyDeviceToHost));

    // Free device memory
    GPU_HANDLE_ERROR(hipFree(d_input));
    GPU_HANDLE_ERROR(hipFree(d_output));
//    GPU_HANDLE_ERROR(hipFree(d_inputInt));
//    GPU_HANDLE_ERROR(hipFree(d_sortedOutput));
    GPU_HANDLE_ERROR(hipFree(d_avg_output));
    GPU_HANDLE_ERROR(hipFree(d_min_output));
    GPU_HANDLE_ERROR(hipFree(d_max_output));
}
