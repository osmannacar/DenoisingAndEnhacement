#include "hip/hip_runtime.h"
#include<stdio.h>
#include "dehaze_kernel.h"
#include "minimum_filter.h"
#include "transpose.h"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""
#include "../Utils.h"

__global__ void d_min_img_kernel(float *src, float *dst, int iWidth, int iHeight, int iChannel){
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	float r, g, b;
	if (id<iHeight*iWidth){
		r = src[id*iChannel + 0];
		g = src[id*iChannel + 1];
		b = src[id*iChannel + 2];
        dst[id] = (r < g) ? r : g;
        dst[id] = (dst[id]< b) ? dst[id] : b;
	}
}

void minfilter(float *d_fog, float *d_min_img, float *d_win_dark, float *d_temp, int width, int height, int channel, int radius){

	dim3 grid1(width / TILE_DIM + 1, height / TILE_DIM + 1);
	dim3 grid2(height / TILE_DIM + 1, width / TILE_DIM + 1);
	dim3 block(TILE_DIM, TILE_DIM);


	if (channel == 3)
	{
		d_min_img_kernel << <(height*width + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE >> >(d_fog, d_min_img, width, height, channel);
	}
	
    d_minfilter_x << <height, BLOCKSIZE>> >(d_min_img, d_temp, width, height, radius);
    transpose << <grid1, block >> >(d_temp, d_min_img, width, height);

    d_minfilter_y << <width, BLOCKSIZE >> >(d_min_img, d_temp, height, width, radius);
    transpose << <grid2, block >> >(d_temp, d_win_dark, height, width);

}
