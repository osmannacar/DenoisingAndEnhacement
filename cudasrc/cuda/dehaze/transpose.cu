#include "hip/hip_runtime.h"
#include "transpose.h"
#include "../Utils.h"

__global__ void transpose(float *idata, float *odata, int width, int height)
{
    __shared__ float tile[TILE_DIM][TILE_DIM+1];

    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
    if (xIndex<width && yIndex<height){
        int index_in = xIndex + (yIndex)*width;
        tile[threadIdx.y][threadIdx.x] = idata[index_in];
    }
    __syncthreads();

    int xIndex_new = blockIdx.y * TILE_DIM + threadIdx.x;
    int yIndex_new = blockIdx.x * TILE_DIM + threadIdx.y;
    if (xIndex_new<height && yIndex_new<width){
        //for (int i=0; i<TILE_DIM; i+=BLOCK_ROWS)
        {
            int index_out = xIndex_new + (yIndex_new)*height;
            //odata[index_out+i*height] = tile[threadIdx.x][threadIdx.y+i];
            odata[index_out] = tile[threadIdx.x][threadIdx.y];
        }
    }
}
