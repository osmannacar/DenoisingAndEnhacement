#include "hip/hip_runtime.h"
#include "dehaze.h"
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""

#include "darkchannel.h"
#include "dehaze_kernel.h"
#include "boxfilter.h"
#include "../Utils.h"


void dehazeGPU(unsigned char *pSrcData, unsigned char * pOutData, int channel, int width, int height){
    float atmos_correct = 0.0;
    float eps = 1e-05f;
    const int tRadius = 7;

    unsigned char   *d_ori;
    unsigned char	*d_dehaze;

    float			*d_fog;
    float           *d_min_img;
    float			*d_win_dark;
    int				*d_index;
    float			*d_im_dark;
    float           *d_dark_mat;
    float			*d_im;
    float			*d_tDown;
    float			*d_foggy_gray;
    float			*d_atmosLight;
    float           *d_atmos;

    float			*d_mean_I;
    float			*d_mean_p;
    float           *d_mean_Ip;
    float           *d_cov_Ip;
    float           *d_mean_II;
    float           *d_var_I;
    float           *d_a;
    float           *d_b;
    float           *d_mean_a;
    float           *d_mean_b;
    float			*d_box_temp;
    float			*d_filtered;
    float			*d_t;
    float			*d_temp;

    GPU_HANDLE_ERROR(hipMalloc((void **)&d_ori, height*width*channel*sizeof(unsigned char)));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_fog, sizeof(float)*width*height*channel));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_dehaze, sizeof(unsigned char)*width*height*channel));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_min_img, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_win_dark, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_im_dark, sizeof(float)*width*height));

    GPU_HANDLE_ERROR(hipMalloc((void **)&d_dark_mat, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_index, sizeof(int)*width*height));//2048 is enough
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_im, sizeof(float)*width*height*channel));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_temp, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_atmosLight, sizeof(float) * 3));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_atmos, sizeof(float) * 3));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_filtered, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_t, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_foggy_gray, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_tDown, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_mean_I, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_mean_p, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_mean_Ip, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_cov_Ip, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_mean_II, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_var_I, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_a, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_b, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_mean_a, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_mean_b, sizeof(float)*width*height));
    GPU_HANDLE_ERROR(hipMalloc((void **)&d_box_temp, sizeof(float)*width*height));

    GPU_HANDLE_ERROR(hipMemcpy(d_ori, pSrcData, sizeof(unsigned char)*width*height*channel, hipMemcpyHostToDevice));

    float_fog_kernel << <(height*width + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE >> >(d_ori, d_fog, d_foggy_gray, width, height, channel);


    minfilter(d_fog, d_min_img, d_win_dark, d_temp, width, height, channel, tRadius);
    MaxReductionkernel << <(height * width ) / 1024 +1, 512 >> >(d_win_dark, d_im_dark, width, height, d_index);//Find 2048 max value into d_index
    MaxReductionkernelTwo << < height * width / 1024 / 512 + 1, 256 >> >(d_im_dark, d_win_dark, d_foggy_gray, width, height, d_index);
    atomsLight_kernel << <1, 1 >> >(d_fog, d_win_dark, d_index, d_im, d_atmosLight, d_atmos, width, height, channel, tRadius);
    atomsLight_kernel_divide << <(height*width + BLOCKSIZE-1) / BLOCKSIZE, BLOCKSIZE >> >(d_fog, d_im, d_atmosLight, width, height, channel);


    minfilter(d_im, d_min_img, d_dark_mat, d_temp, width, height, channel, tRadius);
    t_initial_kernel << <(height*width + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE >> >(d_dark_mat, d_tDown, width, height, channel);

    //Guided Filter

    boxfilter(d_foggy_gray, d_mean_I, d_temp, height, width, tRadius * 5);
    boxfilter(d_tDown, d_mean_p, d_temp, height, width, tRadius * 5);
    matrix_dot_multiple << <(height*width + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE >> >(d_foggy_gray, d_tDown, d_box_temp, height, width);

    boxfilter(d_box_temp, d_mean_Ip, d_temp, height, width, tRadius * 5);
    matrix_dot_multiple << <(height*width + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE >> >(d_foggy_gray, d_foggy_gray, d_box_temp, height, width);

    boxfilter(d_box_temp, d_mean_II, d_temp, height, width, tRadius * 5);
    matrix_cal_a << <(height*width + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE >> >(d_mean_I, d_mean_p, d_mean_II, d_mean_Ip, d_a, eps, height, width);

    matrix_cal_b << <(height*width + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE >> >(d_mean_I, d_mean_p, d_mean_II, d_mean_Ip, d_b, eps, height, width);

    boxfilter(d_a, d_mean_a, d_temp, height, width, tRadius * 5);

    boxfilter(d_b, d_mean_b, d_temp, height, width, tRadius * 5);
    matrix_cal_q << <(height*width + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE >> >(d_mean_a, d_mean_b, d_foggy_gray, d_filtered, width, height);


    clear_kernel << <(height*width + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE >> >(d_fog, d_dehaze, d_atmosLight, d_filtered, width, height, channel, tRadius, atmos_correct);



    GPU_HANDLE_ERROR(hipPeekAtLastError());

    // Wait for all threads to finish
    GPU_HANDLE_ERROR(hipDeviceSynchronize());

    GPU_HANDLE_ERROR(hipMemcpy(pOutData, d_dehaze, width*height*channel * sizeof(unsigned char), hipMemcpyDeviceToHost));


    GPU_HANDLE_ERROR(hipFree(d_ori));
    GPU_HANDLE_ERROR(hipFree(d_fog));
    GPU_HANDLE_ERROR(hipFree(d_dehaze));
    GPU_HANDLE_ERROR(hipFree(d_min_img));
    GPU_HANDLE_ERROR(hipFree(d_win_dark));
    GPU_HANDLE_ERROR(hipFree(d_im_dark));
    GPU_HANDLE_ERROR(hipFree(d_index));
    GPU_HANDLE_ERROR(hipFree(d_dark_mat));
    GPU_HANDLE_ERROR(hipFree(d_im));
    GPU_HANDLE_ERROR(hipFree(d_tDown));
    GPU_HANDLE_ERROR(hipFree(d_foggy_gray));
    GPU_HANDLE_ERROR(hipFree(d_atmos));
    GPU_HANDLE_ERROR(hipFree(d_atmosLight));
    GPU_HANDLE_ERROR(hipFree(d_filtered));
    GPU_HANDLE_ERROR(hipFree(d_t));
    GPU_HANDLE_ERROR(hipFree(d_mean_I));
    GPU_HANDLE_ERROR(hipFree(d_mean_p));
    GPU_HANDLE_ERROR(hipFree(d_mean_Ip));
    GPU_HANDLE_ERROR(hipFree(d_cov_Ip));
    GPU_HANDLE_ERROR(hipFree(d_mean_II));
    GPU_HANDLE_ERROR(hipFree(d_var_I));
    GPU_HANDLE_ERROR(hipFree(d_a));
    GPU_HANDLE_ERROR(hipFree(d_b));
    GPU_HANDLE_ERROR(hipFree(d_mean_a));
    GPU_HANDLE_ERROR(hipFree(d_mean_b));
    GPU_HANDLE_ERROR(hipFree(d_box_temp));
    GPU_HANDLE_ERROR(hipFree(d_temp));
}
