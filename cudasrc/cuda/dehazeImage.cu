#include "hip/hip_runtime.h"
#include "dehazeImage.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "Utils.h"

__global__ void findMinChannel(const unsigned char* image, unsigned char* minChannel, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    unsigned char minVal = 255;

    for (int c = 0; c < channels; c++) {
        int idx = (y * width + x) * channels + c;
        if (image[idx] < minVal) {
            minVal = image[idx];
        }
    }

    minChannel[y * width + x] = minVal;
}

__global__ void boxFilter(const unsigned char* image, float* outputImage, int width, int height, int channels, int kernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    int idx = (y * width + x) * channels;
    float sum = 0.0f;
    int count = 0;

    for (int kx = -kernelSize/2; kx <= kernelSize/2; kx++) {
        for (int ky = -kernelSize/2; ky <= kernelSize/2; ky++) {
            int px = x + kx;
            int py = y + ky;

            if (px < 0 || py < 0 || px >= width || py >= height) {
                continue;
            }

            int pidx = (py * width + px) * channels;
            for (int c = 0; c < channels; c++) {
                sum += image[pidx + c];
            }
            count++;
        }
    }

    outputImage[idx] = sum / count;
    outputImage[idx+1] = sum / count;
    outputImage[idx+2] = sum / count;
}

__global__ void computeTransmission(const unsigned char* minChannel, float* transmission, int width, int height, int kernelSize, float omega) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    int idx = y * width + x;
    float minVal = static_cast<float>(minChannel[idx]) / 255.0f;

    transmission[idx] = 1.0f - omega * minVal;
}

__global__ void dehazeImage(const unsigned char* image, const float* transmission, unsigned char* outputImage, int width, int height, int channels, float tMin) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) {
        return;
    }

    int idx = (y * width + x) * channels;
    float t = fmaxf(transmission[y * width + x], tMin);
    float factor = 1.0f / t;

    for (int c = 0; c < channels; c++) {
        outputImage[idx+c] = fminf(factor * image[idx+c], 255.0f);
    }
}


void cudaDehazeImage(const cv::Mat &input, cv::Mat &output){

    // Create blank output image & calculate size of input and output
    output = cv::Mat(input.size(), input.type());


    unsigned char* imageData = input.data;
    int width = input.cols, height = input.rows, channels = input.channels();

    // Allocate memory for intermediate and output images
    unsigned char* darkChannel = (unsigned char*) malloc(width * height * sizeof(unsigned char));
    float* transmission = (float*) malloc(width * height * sizeof(float));
    unsigned char* outputData = (unsigned char*) malloc(width * height * channels * sizeof(unsigned char));

    // Allocate memory on the GPU
    unsigned char* d_imageData;
    GPU_HANDLE_ERROR(hipMalloc((void**) &d_imageData, width * height * channels * sizeof(unsigned char)));

    unsigned char* d_darkChannel;
    GPU_HANDLE_ERROR(hipMalloc((void**) &d_darkChannel, width * height * sizeof(unsigned char)));

    float* d_transmission;
    GPU_HANDLE_ERROR(hipMalloc((void**) &d_transmission, width * height * sizeof(float)));

    unsigned char* d_outputData;
    GPU_HANDLE_ERROR(hipMalloc((void**) &d_outputData, width * height * channels * sizeof(unsigned char)));

    // Copy input image to GPU
    GPU_HANDLE_ERROR(hipMemcpy(d_imageData, imageData, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice));

    // Compute dark channel
    dim3 blockSize(32, 32);
    dim3 numBlocks((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    findMinChannel<<<numBlocks, blockSize>>>(d_imageData, d_darkChannel, width, height, channels);

    // Compute transmission
    int kernelSize = 5;
    float omega = 0.95f;
    computeTransmission<<<numBlocks, blockSize>>>(d_darkChannel, d_transmission, width, height, kernelSize, omega);

    // Dehaze image
    float tMin = 0.1f;
    dehazeImage<<<numBlocks, blockSize>>>(d_imageData, d_transmission, d_outputData, width, height, channels, tMin);

    // Copy output image back to host
    GPU_HANDLE_ERROR(hipMemcpy(outputData, d_outputData, width * height * channels * sizeof(unsigned char), hipMemcpyDeviceToHost));


    //fill output image
    memcpy(output.data, outputData, width * height * channels * sizeof(unsigned char));

    // Free memory
    free(darkChannel);
    free(transmission);
    free(outputData);
    GPU_HANDLE_ERROR(hipFree(d_imageData));
    GPU_HANDLE_ERROR(hipFree(d_darkChannel));
    GPU_HANDLE_ERROR(hipFree(d_transmission));
    GPU_HANDLE_ERROR(hipFree(d_outputData));

}
